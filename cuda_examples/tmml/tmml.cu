#include "hip/hip_runtime.h"
#include "tmml.hpp"

using namespace std;
using namespace cv;

__constant__ unsigned char const_img_temp_array[TEMPLATE_AREA];

void tmml::cuda_Malloc()
{
    Mat img_work(Size(WORK_WIDTH, WORK_WIDTH), CV_8UC1, Scalar(0));
    img_work_gpu.upload(img_work);
    Mat img_temp(Size(TEMPLATE_WIDTH, TEMPLATE_WIDTH), CV_8UC1, Scalar(0));
    img_temp_gpu.upload(img_temp);
    hipMalloc((void**)& dev_max_val, sizeof(int));
    hipMalloc((void**)&dev_mp, sizeof(Pix));
    hipMalloc((void**)&dev_result_array_bright, sizeof(float) * RESULT_AREA);
} // -- END cuda_Malloc()

void tmml::cuda_Free()
{
    hipFree(&img_work_gpu);
    hipFree(&img_temp_gpu);
    hipFree(dev_max_val);
    hipFree(dev_mp);
    hipFree(dev_result_array_bright);
} // -- END hipFree()

__global__ void match_temp(const cuda::PtrStepSz<unsigned char> img_work_gpu,
                           int * dev_max_val,
                           Pix * dev_v_res_pix,
                           float * dev_result_array_bright)
{
    const int result_id = blockIdx.x * blockDim.x + threadIdx.x;
    int sum_roi_temp = 0;
    int sum_temp_temp = 0;
    int sum_roi_roi = 0;
#ifdef COMBINED
    int sum_roi = 0;
    int sum_temp = 0;
#endif // END ifdef COMBINED
    const int result_y = result_id / RESULT_WIDTH;
    const int result_x = result_id % RESULT_WIDTH;

// 150 mks
    for(int temp_y = 0; temp_y < TEMPLATE_WIDTH; ++temp_y)
    {
        int work_y = temp_y + result_y;
        int temp_id0 = temp_y * TEMPLATE_WIDTH;
        for(int temp_x = 0; temp_x < TEMPLATE_WIDTH; ++temp_x)
        {
            int temp = const_img_temp_array[temp_id0 + temp_x];
            int roi = img_work_gpu(work_y, temp_x + result_x);
            sum_roi_temp += roi * temp;
            sum_temp_temp += temp * temp;
            sum_roi_roi += roi * roi;
#ifdef COMBINED
            sum_roi += roi;
            sum_temp += temp;
#endif // END ifdef COMBINED
        } // for(int tmp_x = 0; tmp_x < TEMPLATE_WIDTH; ++tmp_x)
    } // for(int tmp_y = 0; tmp_y < TEMPLATE_WIDTH; ++tmp_y)

    const float sum_roi_temp1 = TEMPLATE_AREA_1 * sum_roi_temp;
    const float sum_roi_roi1 = TEMPLATE_AREA_1 * sum_roi_roi;
    const float sum_temp_temp1 = TEMPLATE_AREA_1 * sum_temp_temp;
    const float diff_roi_temp2 = sum_roi_roi1 + sum_temp_temp1 - 2.f * sum_roi_temp1;
#ifdef COMBINED
    const float sum_roi1 = TEMPLATE_AREA_1 * sum_roi;
    const float sum_temp1 = TEMPLATE_AREA_1 * sum_temp;
    const float ch  = sum_roi_temp1 - sum_roi1 * sum_temp1;
    const float zn1 = sum_temp_temp1 - sum_temp1 * sum_temp1;
    const float zn2 = sum_roi_roi1 - sum_roi1 * sum_roi1;
    const float result_float = ch / sqrt(zn1 * zn2) -
            KOEFF2LIB_float * diff_roi_temp2 / sqrt(sum_roi_roi1 * sum_temp_temp1);
#endif // END ifdef COMBINED
#ifdef SQDIFF_NORMED
    const float result_float = 1.f - KOEFF2LIB_float * diff_roi_temp2 / sqrt(sum_roi_roi1 * sum_temp_temp1);
#endif // END ifdef SQDIFF_NORMED
//    dev_result_array_bright[result_id] = result_float;
    int val = 1000000 * result_float;
    if(result_id == 0){*dev_max_val = 0;}
    atomicMax(dev_max_val, val);
    __syncthreads();
    if(*dev_max_val == val)
    {
        dev_v_res_pix->x = result_x;
        dev_v_res_pix->y = result_y;
        dev_v_res_pix->bright = *dev_max_val;
    }  // END if(*dev_max_val == val)
}  // END match_temp


void tmml::work_tmml(const Mat& img_work, const Mat& img_temp, Pix& max_pix)
{
    img_work_gpu.upload(img_work);
    hipMemcpyToSymbol(HIP_SYMBOL(const_img_temp_array), img_temp.data, sizeof(unsigned char) * TEMPLATE_AREA);
    //hipMemcpy(dev_max_val, 0, sizeof(int), hipMemcpyHostToDevice);
//    match_temp<<<blocks_match_temp, threads_match_temp>>>(img_work_gpu, dev_max_val, dev_mp, dev_result_array_bright);
    match_temp<<<192, 192>>>(img_work_gpu, dev_max_val, dev_mp, dev_result_array_bright);
    hipMemcpy(&max_pix, dev_mp, sizeof(Pix), hipMemcpyDeviceToHost);
} // END work_tmml
