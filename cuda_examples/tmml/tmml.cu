#include "hip/hip_runtime.h"
﻿#include "tmml.hpp"

using namespace std;
using namespace cv;

__constant__ unsigned char const_img_temp_array[TEMPLATE_AREA];
hipStream_t streamsKernel[numCudaTread];

void tmml::cuda_Malloc()
{
    Mat img_temp(Size(TEMPLATE_WIDTH, TEMPLATE_WIDTH), CV_8UC1, Scalar(0));
    img_temp_gpu.upload(img_temp);

    for(int i = 0; i < numCudaTread; ++i)
    {
        hipMalloc((void**)& dev_mp[i], sizeof(Pix));
    }// END for(int i = 0; i < numCudaTread; ++i)
} // -- END cuda_Malloc()

void tmml::cuda_Free()
{
    hipFree(&img_temp_gpu);
    for(int i = 0; i < numCudaTread; ++i)
    {
        hipFree(&dev_img_work[i]);
        hipFree(dev_mp[i]);
    }// END for(int i = 0; i < numCudaTread; ++i)

} // -- END hipFree()

__global__ void match_temp(const cuda::PtrStepSz<unsigned char> img_work_gpu,
                           Pix * dev_v_res_pix
//                           ,float * dev_result_array_bright
                           )
{
    const int result_id = blockIdx.x * blockDim.x + threadIdx.x;
    int sum_roi_temp = 0;
    int sum_temp_temp = 0;
    int sum_roi_roi = 0;
#ifdef COMBINED
    int sum_roi = 0;
    int sum_temp = 0;
#endif // END ifdef COMBINED
    const int result_y = result_id / RESULT_WIDTH_1;
    const int result_x = result_id % RESULT_WIDTH;

    for(int temp_y = 0; temp_y < TEMPLATE_WIDTH; ++temp_y)
    {
        int work_y = temp_y + result_y;
        int temp_id0 = temp_y * TEMPLATE_WIDTH;
        for(int temp_x = 0; temp_x < TEMPLATE_WIDTH; ++temp_x)
        {
            int temp = const_img_temp_array[temp_id0 + temp_x];
            int roi = img_work_gpu(work_y, temp_x + result_x);
            sum_roi_temp += roi * temp;
            sum_temp_temp += temp * temp;
            sum_roi_roi += roi * roi;
#ifdef COMBINED
            sum_roi += roi;
            sum_temp += temp;
#endif // END ifdef COMBINED
        } // for(int tmp_x = 0; tmp_x < TEMPLATE_WIDTH; ++tmp_x)
    } // for(int tmp_y = 0; tmp_y < TEMPLATE_WIDTH; ++tmp_y)

    const float sum_roi_temp1 = TEMPLATE_AREA_1 * sum_roi_temp;
    const float sum_roi_roi1 = TEMPLATE_AREA_1 * sum_roi_roi;
    const float sum_temp_temp1 = TEMPLATE_AREA_1 * sum_temp_temp;
    const float diff_roi_temp2 = sum_roi_roi1 + sum_temp_temp1 - 2.f * sum_roi_temp1;
#ifdef COMBINED
    const float sum_roi1 = TEMPLATE_AREA_1 * sum_roi;
    const float sum_temp1 = TEMPLATE_AREA_1 * sum_temp;
    const float ch  = sum_roi_temp1 - sum_roi1 * sum_temp1;
    const float zn1 = sum_temp_temp1 - sum_temp1 * sum_temp1;
    const float zn2 = sum_roi_roi1 - sum_roi1 * sum_roi1;
    const float result_float = ch / sqrt(zn1 * zn2) -
            KOEFF2LIB_float * diff_roi_temp2 / sqrt(sum_roi_roi1 * sum_temp_temp1);
#endif // END ifdef COMBINED
#ifdef SQDIFF_NORMED
    const float result_float = 1.f - KOEFF2LIB_float * diff_roi_temp2 / sqrt(sum_roi_roi1 * sum_temp_temp1);
#endif // END ifdef SQDIFF_NORMED

    int val = 1000000 * result_float;
    atomicMax(&dev_v_res_pix->bright, val);
    __syncthreads();
    if(dev_v_res_pix->bright == val)
    {
        dev_v_res_pix->x = result_x;
        dev_v_res_pix->y = result_y;
    }  // END if(*dev_max_val == val)
}  // END match_temp

void tmml::work_tmml(const Mat& img_work, const Mat& img_temp, Pix& max_pix)
{
    hipMemcpyToSymbol(HIP_SYMBOL(const_img_temp_array), img_temp.data, sizeof(unsigned char) * TEMPLATE_AREA);

    for(int i = 0; i < numCudaTread; ++i)
    {
        dev_img_work[i].upload(img_work(Ri[i]), st[i]);
        hipStreamCreate(&streamsKernel[i]);
        match_temp<<<blocks, threads, 0, streamsKernel[i]>>>(dev_img_work[i], dev_mp[i] );
    }// END for(int i = 0; i < numCudaTread; ++i)

    for(int i = 0; i < numCudaTread; ++i)
    {
        hipMemcpy(&host_mp[i], dev_mp[i], sizeof(Pix), hipMemcpyDeviceToHost);
    }// END for(int i = 0; i < numCudaTread; ++i)

    max_pix = maxValue(host_mp);

} // END work_tmml

Pix tmml::maxValue(Pix host_mp[numCudaTread])
{
    Pix max;
    max.bright = 0;
    for(int i = 0; i < numCudaTread; ++i)
    {
        if(host_mp[i].bright > max.bright) max = host_mp[i];
    }
    return  max;
} // END maxValue


