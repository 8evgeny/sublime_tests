#include "hip/hip_runtime.h"
﻿#include "tmml.hpp"

using namespace std;
using namespace cv;

__constant__ unsigned char const_img_temp_array[TEMPLATE_AREA];

void tmml::cuda_Malloc()
{
    Mat img_work_1(Size(96, 240), CV_8UC1, Scalar(0));
    Mat img_work_2(Size(96, 240), CV_8UC1, Scalar(0));
    Mat img_work_3(Size(96, 240), CV_8UC1, Scalar(0));
    Mat img_work_4(Size(96, 240), CV_8UC1, Scalar(0));
    img_work_gpu[0].upload(img_work_1);
    img_work_gpu[1].upload(img_work_2);
    img_work_gpu[2].upload(img_work_3);
    img_work_gpu[3].upload(img_work_4);

    Mat img_temp(Size(TEMPLATE_WIDTH, TEMPLATE_WIDTH), CV_8UC1, Scalar(0));
    img_temp_gpu.upload(img_temp);

    for(int i = 0; i < numCudaTread; ++i)
    {
        hipMalloc((void**)& dev_max_val[i], sizeof(int));
        hipMalloc((void**)& dev_mp[i], sizeof(Pix *));
    }// END for(int i = 0; i < numCudaTread; ++i)

//    hipMalloc((void**)&dev_result_array_bright, sizeof(float) * RESULT_AREA);
} // -- END cuda_Malloc()

void tmml::cuda_Free()
{
    hipFree(&img_temp_gpu);
    for(int i = 0; i < numCudaTread; ++i)
    {
        hipFree(&img_work_gpu[i]);
        hipFree(dev_max_val[i]);
        hipFree(dev_mp[i]);
    }// END for(int i = 0; i < numCudaTread; ++i)

//    hipFree(dev_result_array_bright);
} // -- END hipFree()

__global__ void match_temp(const cuda::PtrStepSz<unsigned char> img_work_gpu,
                           Pix * dev_v_res_pix
//                           ,float * dev_result_array_bright
                           )
{
    const int result_id = blockIdx.x * blockDim.x + threadIdx.x;
    int sum_roi_temp = 0;
    int sum_temp_temp = 0;
    int sum_roi_roi = 0;
#ifdef COMBINED
    int sum_roi = 0;
    int sum_temp = 0;
#endif // END ifdef COMBINED
    const int result_y = result_id / 48; //96 - 48
    const int result_x = result_id % 192;//240 - 48

// 150 mks
    for(int temp_y = 0; temp_y < TEMPLATE_WIDTH; ++temp_y)
    {
        int work_y = temp_y + result_y;
        int temp_id0 = temp_y * TEMPLATE_WIDTH;
        for(int temp_x = 0; temp_x < TEMPLATE_WIDTH; ++temp_x)
        {
            int temp = const_img_temp_array[temp_id0 + temp_x];
            int roi = img_work_gpu(work_y, temp_x + result_x);
            sum_roi_temp += roi * temp;
            sum_temp_temp += temp * temp;
            sum_roi_roi += roi * roi;
#ifdef COMBINED
            sum_roi += roi;
            sum_temp += temp;
#endif // END ifdef COMBINED
        } // for(int tmp_x = 0; tmp_x < TEMPLATE_WIDTH; ++tmp_x)
    } // for(int tmp_y = 0; tmp_y < TEMPLATE_WIDTH; ++tmp_y)

    const float sum_roi_temp1 = TEMPLATE_AREA_1 * sum_roi_temp;
    const float sum_roi_roi1 = TEMPLATE_AREA_1 * sum_roi_roi;
    const float sum_temp_temp1 = TEMPLATE_AREA_1 * sum_temp_temp;
    const float diff_roi_temp2 = sum_roi_roi1 + sum_temp_temp1 - 2.f * sum_roi_temp1;
#ifdef COMBINED
    const float sum_roi1 = TEMPLATE_AREA_1 * sum_roi;
    const float sum_temp1 = TEMPLATE_AREA_1 * sum_temp;
    const float ch  = sum_roi_temp1 - sum_roi1 * sum_temp1;
    const float zn1 = sum_temp_temp1 - sum_temp1 * sum_temp1;
    const float zn2 = sum_roi_roi1 - sum_roi1 * sum_roi1;
    const float result_float = ch / sqrt(zn1 * zn2) -
            KOEFF2LIB_float * diff_roi_temp2 / sqrt(sum_roi_roi1 * sum_temp_temp1);
#endif // END ifdef COMBINED
#ifdef SQDIFF_NORMED
    const float result_float = 1.f - KOEFF2LIB_float * diff_roi_temp2 / sqrt(sum_roi_roi1 * sum_temp_temp1);
#endif // END ifdef SQDIFF_NORMED
//    dev_result_array_bright[result_id] = result_float;
    int val = 1000000 * result_float;
//    if(result_id == 0){*dev_max_val = 0;}
    atomicMax(&dev_v_res_pix->bright, val);
    __syncthreads();
    if(dev_v_res_pix->bright == val)
    {
        dev_v_res_pix->x = result_x;
        dev_v_res_pix->y = result_y;
    }  // END if(*dev_max_val == val)
}  // END match_temp

void tmml::work_tmml(const Mat& img_work, const Mat& img_temp, Pix& max_pix)
{
    hipMemcpyToSymbol(HIP_SYMBOL(const_img_temp_array), img_temp.data, sizeof(unsigned char) * TEMPLATE_AREA);
    hipStream_t *streamsKernel = (hipStream_t *)malloc(numCudaTread * sizeof(hipStream_t));
    for (int i = 0 ; i < numCudaTread; i++)
    {
        hipStreamCreate(&streamsKernel[i]);
    }

//    for(int i = 0; i < numCudaTread; ++i)
//    {
    int i = 0;
    img_work_gpu[i].upload(img_work(Ri[i]), st[i]);
    match_temp<<<blocks, threads, 0, streamsKernel[i]>>>(img_work_gpu[i], dev_mp[i] );
    i = 1;
    img_work_gpu[i].upload(img_work(Ri[i]), st[i]);
    match_temp<<<blocks, threads, 0, streamsKernel[i]>>>(img_work_gpu[i], dev_mp[i] );
    i = 2;
    img_work_gpu[i].upload(img_work(Ri[i]), st[i]);
    match_temp<<<blocks, threads, 0, streamsKernel[i]>>>(img_work_gpu[i], dev_mp[i] );
    i = 3;
    img_work_gpu[i].upload(img_work(Ri[i]), st[i]);
    match_temp<<<blocks, threads, 0, streamsKernel[i]>>>(img_work_gpu[i], dev_mp[i] );
//    }
    hipMemcpy(&max_pix, maxValue(dev_mp), sizeof(Pix), hipMemcpyDeviceToHost);
} // END work_tmml

Pix * tmml::maxValue(Pix * dev_mp[numCudaTread])
{
//    int num = 0;
//    a->bright>b->bright ? num = 0 : b->bright>c->bright ? num = 1 : num = 2;


    return dev_mp[1];
} // END maxValue


