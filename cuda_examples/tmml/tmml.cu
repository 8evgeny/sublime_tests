#include "hip/hip_runtime.h"
﻿#include "tmml.hpp"
#include <thread>

using namespace std;
using namespace cv;

__constant__ unsigned char const_img_temp_array[TEMPLATE_AREA];

void tmml::cuda_Malloc()
{
    Mat img_work_1(Size(WORK_WIDTH, 60), CV_8UC1, Scalar(0));
    Mat img_work_2(Size(WORK_WIDTH, 60), CV_8UC1, Scalar(0));
    Mat img_work_3(Size(WORK_WIDTH, 60), CV_8UC1, Scalar(0));
    Mat img_work_4(Size(WORK_WIDTH, 59), CV_8UC1, Scalar(0));
    img_work_gpu_1.upload(img_work_1);
    img_work_gpu_2.upload(img_work_2);
    img_work_gpu_3.upload(img_work_3);
    img_work_gpu_4.upload(img_work_4);

    Mat img_temp(Size(TEMPLATE_WIDTH, TEMPLATE_WIDTH), CV_8UC1, Scalar(0));
    img_temp_gpu.upload(img_temp);
    hipMalloc((void**)& dev_max_val_1, sizeof(int));
    hipMalloc((void**)& dev_max_val_2, sizeof(int));
    hipMalloc((void**)& dev_max_val_3, sizeof(int));
    hipMalloc((void**)& dev_max_val_4, sizeof(int));

    hipMalloc((void**)&dev_mp_1, sizeof(Pix));
    hipMalloc((void**)&dev_mp_2, sizeof(Pix));
    hipMalloc((void**)&dev_mp_3, sizeof(Pix));
    hipMalloc((void**)&dev_mp_4, sizeof(Pix));

//    hipMalloc((void**)&dev_result_array_bright, sizeof(float) * RESULT_AREA);
} // -- END cuda_Malloc()

void tmml::cuda_Free()
{
    hipFree(&img_work_gpu_1);
    hipFree(&img_work_gpu_2);
    hipFree(&img_work_gpu_3);
    hipFree(&img_work_gpu_4);

    hipFree(&img_temp_gpu);
    hipFree(dev_max_val_1);
    hipFree(dev_max_val_2);
    hipFree(dev_max_val_3);
    hipFree(dev_max_val_4);

    hipFree(dev_mp_1);
    hipFree(dev_mp_2);
    hipFree(dev_mp_3);
    hipFree(dev_mp_4);
//    hipFree(dev_result_array_bright);
} // -- END hipFree()

__global__ void match_temp(const cuda::PtrStepSz<unsigned char> img_work_gpu,
                           int * dev_max_val,
                           Pix * dev_v_res_pix
//                           ,float * dev_result_array_bright
                           )
{
    const int result_id = blockIdx.x * blockDim.x + threadIdx.x;
    int sum_roi_temp = 0;
    int sum_temp_temp = 0;
    int sum_roi_roi = 0;
#ifdef COMBINED
    int sum_roi = 0;
    int sum_temp = 0;
#endif // END ifdef COMBINED
    const int result_y = result_id / 72; //120 - 48
    const int result_x = result_id % 72;

// 150 mks
    for(int temp_y = 0; temp_y < TEMPLATE_WIDTH; ++temp_y)
    {
        int work_y = temp_y + result_y;
        int temp_id0 = temp_y * TEMPLATE_WIDTH;
        for(int temp_x = 0; temp_x < TEMPLATE_WIDTH; ++temp_x)
        {
            int temp = const_img_temp_array[temp_id0 + temp_x];
            int roi = img_work_gpu(work_y, temp_x + result_x);
            sum_roi_temp += roi * temp;
            sum_temp_temp += temp * temp;
            sum_roi_roi += roi * roi;
#ifdef COMBINED
            sum_roi += roi;
            sum_temp += temp;
#endif // END ifdef COMBINED
        } // for(int tmp_x = 0; tmp_x < TEMPLATE_WIDTH; ++tmp_x)
    } // for(int tmp_y = 0; tmp_y < TEMPLATE_WIDTH; ++tmp_y)

    const float sum_roi_temp1 = TEMPLATE_AREA_1 * sum_roi_temp;
    const float sum_roi_roi1 = TEMPLATE_AREA_1 * sum_roi_roi;
    const float sum_temp_temp1 = TEMPLATE_AREA_1 * sum_temp_temp;
    const float diff_roi_temp2 = sum_roi_roi1 + sum_temp_temp1 - 2.f * sum_roi_temp1;
#ifdef COMBINED
    const float sum_roi1 = TEMPLATE_AREA_1 * sum_roi;
    const float sum_temp1 = TEMPLATE_AREA_1 * sum_temp;
    const float ch  = sum_roi_temp1 - sum_roi1 * sum_temp1;
    const float zn1 = sum_temp_temp1 - sum_temp1 * sum_temp1;
    const float zn2 = sum_roi_roi1 - sum_roi1 * sum_roi1;
    const float result_float = ch / sqrt(zn1 * zn2) -
            KOEFF2LIB_float * diff_roi_temp2 / sqrt(sum_roi_roi1 * sum_temp_temp1);
#endif // END ifdef COMBINED
#ifdef SQDIFF_NORMED
    const float result_float = 1.f - KOEFF2LIB_float * diff_roi_temp2 / sqrt(sum_roi_roi1 * sum_temp_temp1);
#endif // END ifdef SQDIFF_NORMED
//    dev_result_array_bright[result_id] = result_float;
    int val = 1000000 * result_float;
    if(result_id == 0){*dev_max_val = 0;}
    atomicMax(dev_max_val, val);
    __syncthreads();
    if(*dev_max_val == val)
    {
        dev_v_res_pix->x = result_x;
        dev_v_res_pix->y = result_y;
        dev_v_res_pix->bright = *dev_max_val;
    }  // END if(*dev_max_val == val)
}  // END match_temp


void tmml::work_tmml(const Mat& img_work, const Mat& img_temp, Pix& max_pix)
{
    hipMemcpyToSymbol(HIP_SYMBOL(const_img_temp_array), img_temp.data, sizeof(unsigned char) * TEMPLATE_AREA);

//    thread t1([&]{
//        img_work_gpu_1.upload(img_work(Range(0, 119), Range(0, 119)));
//        match_temp<<<48, 48>>>(img_work_gpu_1, dev_max_val_1, dev_mp_1 );
//    });
//    thread t2([&]{
//        img_work_gpu_2.upload(img_work(Range(0, 119), Range(120, 239)));
//        match_temp<<<48, 48>>>(img_work_gpu_2, dev_max_val_2, dev_mp_2 );
//    });
//    thread t3([&]{
//        img_work_gpu_3.upload(img_work(Range(120, 239), Range(0, 119)));
//        match_temp<<<48, 48>>>(img_work_gpu_3, dev_max_val_3, dev_mp_3 );
//    });
//    thread t4([&]{
//        img_work_gpu_4.upload(img_work(Range(120, 239), Range(120, 239)));
//        match_temp<<<48, 48>>>(img_work_gpu_4, dev_max_val_4, dev_mp_4 );
//    });
//t1.join();
//t2.join();
//t3.join();
//t4.join();

        img_work_gpu_1.upload(img_work(Range(0, 119), Range(0, 119)));
        match_temp<<<48, 48>>>(img_work_gpu_1, dev_max_val_1, dev_mp_1 );

        img_work_gpu_2.upload(img_work(Range(0, 119), Range(120, 239)));
        match_temp<<<48, 48>>>(img_work_gpu_2, dev_max_val_2, dev_mp_2 );

        img_work_gpu_3.upload(img_work(Range(120, 239), Range(0, 119)));
        match_temp<<<48, 48>>>(img_work_gpu_3, dev_max_val_3, dev_mp_3 );

        img_work_gpu_4.upload(img_work(Range(120, 239), Range(120, 239)));
        match_temp<<<48, 48>>>(img_work_gpu_4, dev_max_val_4, dev_mp_4 );

    hipMemcpy(&max_pix, dev_mp_4, sizeof(Pix), hipMemcpyDeviceToHost);

} // END work_tmml

