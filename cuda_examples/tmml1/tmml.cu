#include "hip/hip_runtime.h"
﻿#include "tmml.hpp"

using namespace std;
using namespace cv;

__constant__ unsigned char const_img_temp_array[TEMPLATE_AREA];
hipStream_t streamsKernel[numCudaTread];

void tmml::cuda_Malloc()
{
    Mat img_work(Size(WORK_WIDTH, WORK_WIDTH), CV_8UC1, Scalar(0));
    img_work_gpu.upload(img_work);
    Mat img_temp(Size(TEMPLATE_WIDTH, TEMPLATE_WIDTH), CV_8UC1, Scalar(0));
    img_temp_gpu.upload(img_temp);
    for(int i = 0; i < numCudaTread; ++i)
    {
        hipMalloc((void**)& dev_mp[i], sizeof(Pix));
    }// END for(int i = 0; i < numCudaTread; ++i)
} // -- END cuda_Malloc()

void tmml::cuda_Free()
{
    hipFree(&img_work_gpu);
    hipFree(&img_temp_gpu);
    for(int i = 0; i < numCudaTread; ++i)
    {
        hipFree(&dev_img_work[i]);
        hipFree(dev_mp[i]);
    }// END for(int i = 0; i < numCudaTread; ++i)
} // -- END hipFree()

__global__ void match_temp(const cuda::PtrStepSz<unsigned char> img_work_gpu,
                           Pix * dev_v_res_pix,
                           int i
                           )
{
    const int result_id = blockIdx.x * blockDim.x + threadIdx.x + i * RESULT_AREA_n;
    int sum_roi_temp = 0;
    int sum_temp_temp = 0;
    int sum_roi_roi = 0;
#ifdef COMBINED
    int sum_roi = 0;
    int sum_temp = 0;
#endif // END ifdef COMBINED
    const int result_y = result_id / RESULT_WIDTH;
    const int result_x = result_id % RESULT_WIDTH;
    for(int temp_y = 0; temp_y < TEMPLATE_WIDTH; ++temp_y)
    {
        int work_y = temp_y + result_y;
        int temp_id0 = temp_y * TEMPLATE_WIDTH;
        for(int temp_x = 0; temp_x < TEMPLATE_WIDTH; ++temp_x)
        {
            int temp = const_img_temp_array[temp_id0 + temp_x];
            int roi = img_work_gpu(work_y, temp_x + result_x);
            sum_roi_temp += roi * temp;
            sum_temp_temp += temp * temp;
            sum_roi_roi += roi * roi;
#ifdef COMBINED
            sum_roi += roi;
            sum_temp += temp;
#endif // END ifdef COMBINED
        } // for(int tmp_x = 0; tmp_x < TEMPLATE_WIDTH; ++tmp_x)
    } // for(int tmp_y = 0; tmp_y < TEMPLATE_WIDTH; ++tmp_y)
    const float sum_roi_temp1 = TEMPLATE_AREA_1 * sum_roi_temp;
    const float sum_roi_roi1 = TEMPLATE_AREA_1 * sum_roi_roi;
    const float sum_temp_temp1 = TEMPLATE_AREA_1 * sum_temp_temp;
    const float diff_roi_temp2 = sum_roi_roi1 + sum_temp_temp1 - 2.f * sum_roi_temp1;
#ifdef COMBINED
    const float sum_roi1 = TEMPLATE_AREA_1 * sum_roi;
    const float sum_temp1 = TEMPLATE_AREA_1 * sum_temp;
    const float ch  = sum_roi_temp1 - sum_roi1 * sum_temp1;
    const float zn1 = sum_temp_temp1 - sum_temp1 * sum_temp1;
    const float zn2 = sum_roi_roi1 - sum_roi1 * sum_roi1;
    const float result_float = ch / sqrt(zn1 * zn2) -
            KOEFF2LIB_float * diff_roi_temp2 / sqrt(sum_roi_roi1 * sum_temp_temp1);
#endif // END ifdef COMBINED
#ifdef SQDIFF_NORMED
    const float result_float = 1.f - KOEFF2LIB_float * diff_roi_temp2 / sqrt(sum_roi_roi1 * sum_temp_temp1);
#endif // END ifdef SQDIFF_NORMED
    int val = 1000000 * result_float;
    atomicMax(&dev_v_res_pix->bright, val);
    __syncthreads();
    if(dev_v_res_pix->bright == val)
    {
        dev_v_res_pix->x = result_x;
        dev_v_res_pix->y = result_y;
    }  // END if(*dev_max_val == val)
}  // END match_temp

void tmml::thr_gpu(int i, const cuda::PtrStepSz<unsigned char> img_work_gpu, Pix * host_mp)
{
    hipStreamCreate(&streamsKernel[i]);
    match_temp<<<blocks_match_temp, threads_match_temp, 0, streamsKernel[i]>>>(img_work_gpu, dev_mp[i], i);
    //match_temp<<<blocks_match_temp, threads_match_temp>>>(img_work_gpu, dev_mp[i], i);
    hipMemcpy(&host_mp[i], dev_mp[i], sizeof(Pix), hipMemcpyDeviceToHost);
} // END thr_gpu

void tmml::work_tmml(const Mat& img_work, const Mat& img_temp, Pix& max_pix)
{
    img_work_gpu.upload(img_work);
    hipMemcpyToSymbol(HIP_SYMBOL(const_img_temp_array), img_temp.data, sizeof(unsigned char) * TEMPLATE_AREA);
    for(int i = 0; i < numCudaTread; ++i)
    {
        //arr_th[i] = thread(&tmml::thr_gpu, this, i, img_work_gpu, ref(host_mp));
        //if(i == numCudaTread_1){arr_th[i].join();}
        //else{arr_th[i].detach();}
        thr_gpu(i, img_work_gpu, host_mp);
    } // END for(int i = 0; i < numCudaTread; ++i)

//    for(int i = 0; i < numCudaTread; ++i)
//    {
//        arr_th[i].join();
//    }// END for(int i = 0; i < numCudaTread; ++i)
    maxValue(host_mp, max_pix);
} // END work_tmml

void tmml::maxValue(Pix * host_mp, Pix& max)
{
    int maxIndex = 0;
    int bright = 0;
    for(int i = 0; i < numCudaTread; ++i)
    {
        int bright_i = host_mp[i].bright;
        if(bright_i > bright)
        {
            maxIndex = i;
            bright = bright_i;
        } // END if(bright_i > bright)
    } // END for(int i = 0; i < numCudaTread; ++i)
    max = host_mp[maxIndex];
} // END maxValue


