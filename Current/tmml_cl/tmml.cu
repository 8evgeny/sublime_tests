#include "hip/hip_runtime.h"
#include "tmml.hpp"

using namespace std;
using namespace cv;

__constant__ unsigned char const_img_temp_array[TEMPLATE_AREA];
__constant__ unsigned char const_first_level_x[N1];
__constant__ unsigned char const_first_level_y[N1];

void tmml::cuda_Malloc()
{
    hipMalloc((void**)&dev_result_array_x, sizeof(unsigned char) * RESULT_AREA);
    hipMalloc((void**)&dev_result_array_y, sizeof(unsigned char) * RESULT_AREA);
    hipMalloc((void**)&dev_result_array_bright, sizeof(float) * RESULT_AREA);
    Mat img_work(Size(WORK_WIDTH, WORK_HEIGHT), CV_8UC1, Scalar(0));
    img_work_gpu.upload(img_work);
    Mat img_temp(Size(TEMPLATE_WIDTH, TEMPLATE_HEIGHT), CV_8UC1, Scalar(0));
    img_temp_gpu.upload(img_temp);
    hipMalloc((void**)&dev_max_K1, sizeof(Pix) * N1);
    hipMalloc((void**)&dev_max_K2, sizeof(Pix) * N2);
} // -- END cuda_Malloc()

void tmml::cuda_Free()
{
    hipFree(&img_work_gpu);
    hipFree(&img_temp_gpu);
    hipFree(dev_result_array_x);
    hipFree(dev_result_array_y);
    hipFree(dev_result_array_bright);
    hipFree(dev_max_K1);
    hipFree(dev_max_K2);
    hipFree(const_img_temp_array);
    hipFree(const_first_level_x);
    hipFree(const_first_level_y);
} // -- END hipFree()

void tmml::fill_result_array()
{
    int id = 0;
    for(int y = 0; y < RESULT_HEIGHT; ++y)
    {
        for(int x = 0; x < RESULT_WIDTH; ++x)
        {
            result_array_x[id] = (unsigned char)x;
            result_array_y[id] = (unsigned char)y;
            id++;
        } // END for(int x = 0; x < RESULT_WIDTH; ++x)
    } // END for(int y = 0; y < RESULT_HEIGHT; ++y)
    hipMemcpy(dev_result_array_x, result_array_x, sizeof(unsigned char) * RESULT_AREA, hipMemcpyHostToDevice);
    hipMemcpy(dev_result_array_y, result_array_y, sizeof(unsigned char) * RESULT_AREA, hipMemcpyHostToDevice);
} // END fill_result_array

void tmml::fill_template_array()
{
    int id = 0;
    for(int y = 0; y < TEMPLATE_HEIGHT; ++y)
    {
        for(int x = 0; x < TEMPLATE_WIDTH; ++x)
        {
            template_array_x[id] = (unsigned char)x;
            template_array_y[id] = (unsigned char)y;
            id++;
        }  // END for(int x = 0; x < TEMPLATE_WIDTH; ++x)
    }  // END for(int y = 0; y < TEMPLATE_HEIGHT; ++y)
    hipMemcpy(dev_template_array_x, template_array_x, sizeof(unsigned char) * TEMPLATE_AREA, hipMemcpyHostToDevice);
    hipMemcpy(dev_template_array_y, template_array_y, sizeof(unsigned char) * TEMPLATE_AREA, hipMemcpyHostToDevice);
} // END fill_template_array

void tmml::fill1level()
{       
    int id = 0;
    for(int y = 0; y < RESULT_HEIGHT; ++y)
    {
        for(int x = 0; x < RESULT_WIDTH; x += K1)
        {
            first_level_x[id] = (unsigned char)x;
            first_level_y[id] = (unsigned char)y;
            id++;
        } // END for(int x = 0; x < RESULT_WIDTH; x += K1)
    } // END for(int x = 0; x < RESULT_WIDTH; x += K1)
    hipMemcpyToSymbol(HIP_SYMBOL(const_first_level_x), first_level_x, sizeof(unsigned char) * N1);
    hipMemcpyToSymbol(HIP_SYMBOL(const_first_level_y), first_level_y, sizeof(unsigned char) * N1);
} // END fill1level

#ifdef SQDIFF_NORMED
__global__ void match_temp(const cuda::PtrStepSz<unsigned char> img_work_gpu, float * dev_result_array_bright)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int sum_roi_temp = 0;
    int diff_roi_temp = 0;
    int y = id / RESULT_WIDTH;
    int x = id % RESULT_WIDTH;
    for(int tmp_y = 0; tmp_y < TEMPLATE_HEIGHT; ++tmp_y)
    {
        for(int tmp_x = 0; tmp_x < TEMPLATE_WIDTH; ++tmp_x)
        {
            int i = tmp_y * TEMPLATE_WIDTH + tmp_x;
            unsigned char temp = const_img_temp_array[i];
            unsigned char roi = img_work_gpu(tmp_y + y, tmp_x + x);
            diff_roi_temp += abs(roi - temp);
            sum_roi_temp += (roi + temp);
        } // for(int tmp_x = 0; tmp_x < TEMPLATE_WIDTH; ++tmp_x)
    } // for(int tmp_y = 0; tmp_y < TEMPLATE_HEIGHT; ++tmp_y)
    dev_result_array_bright[id] = 1.f - (float)diff_roi_temp / ((float)sum_roi_temp);
}  // END void match_temp
#endif // END #ifdef SQDIFF_NORMED

#ifdef CCOEFF_NORMED
__global__ void match_temp(const cuda::PtrStepSz<unsigned char> img_work_gpu, float * dev_result_array_bright)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int sum_roi_temp = 0;
    int sum_temp_temp = 0;
    int sum_roi_roi = 0;
    int sum_roi = 0;
    int sum_temp = 0;
    int y = id / RESULT_WIDTH;
    int x = id % RESULT_WIDTH;
    for(int tmp_y = 0; tmp_y < TEMPLATE_HEIGHT; ++tmp_y)
    {
        for(int tmp_x = 0; tmp_x < TEMPLATE_WIDTH; ++tmp_x)
        {
            int i = tmp_y * TEMPLATE_WIDTH + tmp_x;
            unsigned char temp = const_img_temp_array[i];
            unsigned char roi = img_work_gpu(tmp_y + y, tmp_x + x);
            sum_roi_temp += roi * temp;
            sum_temp_temp += temp * temp;
            sum_roi_roi += roi * roi;
            sum_roi += roi;
            sum_temp += temp;
        } // for(int tmp_x = 0; tmp_x < TEMPLATE_WIDTH; ++tmp_x)
    } // for(int tmp_y = 0; tmp_y < TEMPLATE_HEIGHT; ++tmp_y)
    const long long ch  = (long long)TEMPLATE_AREA * sum_roi_temp - (long long)sum_roi * sum_temp;
    const long long zn1 = (long long)TEMPLATE_AREA * sum_temp_temp - (long long)sum_temp * sum_temp;
    const long long zn2 = (long long)TEMPLATE_AREA * sum_roi_roi - (long long)sum_roi * sum_roi;
    const double sq1 = sqrt((double)zn1);
    const double sq2 = sqrt((double)zn2);
    dev_result_array_bright[id] = (double)ch / (sq1 * sq2);
}  // END void match_temp
#endif // END #ifdef CCOEFF_NORMED

#ifdef COMBINED
__global__ void match_temp(const cuda::PtrStepSz<unsigned char> img_work_gpu, float * dev_result_array_bright)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int sum_roi_temp = 0;
    int sum_temp_temp = 0;
    int sum_roi_roi = 0;
    int sum_roi = 0;u, &minVal, &maxVal, &minLoc, &maxLoc, Mat());
    if(maxLoc.x != temp_left || maxLoc.y != temp_top){cout << "CPU iter=" << iter << " !!!" << endl; break;}
}  // END for(int iter = 0; iter < iter_num; ++iter)
time_end = system_clock::now();
duration_matching = time_end - time_start;
cout.precision(2);
std::cout.setf(std::ios::fixed);
cout << "Duration OpenCV =\t" << 1e3 * duration_matching.count()/iter_num << " ms" << endl;

//CUDA
time_start = system_clock::now();

for(int iter = 0; iter < iter_num; ++iter)
{
    tm->work_cuda(img_work, img_temp, tm->max_pix);
    if(tm->max_pix.x != temp_left || tm->max_pix.y != temp_top){cout << "GPU iter=" << iter << " !!!" << endl; break;}
}  // END for(int iter = 0; iter < iter_num; ++iter)
time_end = system_clock::now();
duration_matching = time_end - time_start;
cout << "Duration CUDA =\t\t" << 1e3 * duration_matching.count()/iter_num  << " ms" << endl;
cout << "cuda xy =\t\t[" << (int)tm->max_pix.x << ", " << (int)tm->max_pix.y << "] " /*<<"   bright= " << tm->max_pix.bright*/ << endl;

//OpenCL
matchingOpenCL(img_work, img_temp);

//Results
tm->fill_result_array();
double sum_diff = 0;
for(int id = 0; id < RESULT_AREA; id++)
{
    int x = tm->result_array_x[id];
    int y = tm->result_array_y[id];
    float bright_gpu = tm->result_array_bright[id];
    float bright_cpu = img_result_cpu.at<float>(y, x);
    img_result_cuda.at<float>(y, x) = bright_gpu;
    double diff = abs(bright_cpu - bright_gpu);
    //cout << id << "; x = " << x << "; y = " << y << "; cpu = " << bright_cpu << "; gpu = " << bright_gpu << endl;
    sum_diff += diff;
} // END for(int id = 0; id < RESULT_AREA; id++)
//    cout << "sum_diff = " << sum_diff << "; RESULT_AREA = " << RESULT_AREA << "; raitio = " << sum_diff/RESULT_AREA << endl;


normalize(img_result_cpu, img_result_cpu, 0, 255, NORM_MINMAX);
img_result_cpu.convertTo(img_result_cpu, CV_8UC1);
int k = 2;
resize(img_result_cpu, img_result_cpu, Size(k*RESULT_WIDTH, k*RESULT_HEIGHT));
const char* OpenCV_window = "OpenCV";
namedWindow( OpenCV_window, WINDOW_AUTOSIZE );
moveWindow(OpenCV_window, 900,100);
imshow(OpenCV_window, img_result_cpu);

normalize(img_result_cuda, img_result_cuda, 0, 255, NORM_MINMAX);

cout<<"CUDA"<<endl;
for (int i = res.xpos  + res.ypos * (img_result_cuda.cols - img_temp.cols + 1);
     i < res.xpos + res.ypos * (img_result_cuda.cols - img_temp.cols + 1) + 10; ++i)
{
    cout<<img_result_cuda.at<uint>(i)<<"  ";
}
cout<<endl;

img_result_cuda.convertTo(img_result_cuda, CV_8UC1);
resize(img_result_cuda, img_result_cuda, Size(k*RESULT_WIDTH, k*RESULT_HEIGHT));
const char* CUDA_window = "CUDA";
namedWindow( CUDA_window, WINDOW_AUTOSIZE );
moveWindow(CUDA_window, 1300,100);
imshow(CUDA_window, img_result_cuda);

normalize(img_result_CL, img_result_CL, 0, 255, NORM_MINMAX);
img_result_CL.convertTo(img_result_CL, CV_8UC1);
resize(img_result_CL, img_result_CL, Size(k*RESULT_WIDTH, k*RESULT_HEIGHT));
const char* CL_window = "OpenCL";
namedWindow( CL_window, WINDOW_AUTOSIZE );
moveWindow(CL_window, 900,600);
imshow(CL_window, img_result_CL);

cv::cvtColor(img_work,img_work,cv::COLOR_GRAY2BGR);
cv::rectangle(img_work, cv::Point(res.xpos, res.ypos), cv::Point(res.xpos+img_temp.cols, res.ypos+img_temp.rows),cv::Scalar(0,0,255),3);
const char* OpenCL = "matchingOpenCL";
namedWindow( OpenCL, WINDOW_AUTOSIZE );
moveWindow(OpenCL, 1300,600);
resize(img_work, img_work, Size(k*RESULT_WIDTH, k*RESULT_HEIGHT));
imshow(OpenCL, img_work);

unsigned char key = waitKey(0);
tm.reset();
return 0;
} // END main

    int sum_temp = 0;
    int sum_roi_temp_2 = 0;
    int diff_roi_temp = 0;
    int y = id / RESULT_WIDTH;
    int x = id % RESULT_WIDTH;
    for(int tmp_y = 0; tmp_y < TEMPLATE_HEIGHT; ++tmp_y)
    {
        for(int tmp_x = 0; tmp_x < TEMPLATE_WIDTH; ++tmp_x)
        {
            int i = tmp_y * TEMPLATE_WIDTH + tmp_x;
            unsigned char temp = const_img_temp_array[i];
            unsigned char roi = img_work_gpu(tmp_y + y, tmp_x + x);
            sum_roi_temp += roi * temp;
            sum_temp_temp += temp * temp;
            sum_roi_roi += roi * roi;
            sum_roi += roi;
            sum_temp += temp;
            diff_roi_temp += abs(roi - temp);
            sum_roi_temp_2 += (roi + temp);
        } // for(int tmp_x = 0; tmp_x < TEMPLATE_WIDTH; ++tmp_x)
    } // for(int tmp_y = 0; tmp_y < TEMPLATE_HEIGHT; ++tmp_y)
    const long long ch  = (long long)TEMPLATE_AREA * sum_roi_temp - (long long)sum_roi * sum_temp;
    const long long zn1 = (long long)TEMPLATE_AREA * sum_temp_temp - (long long)sum_temp * sum_temp;
    const long long zn2 = (long long)TEMPLATE_AREA * sum_roi_roi - (long long)sum_roi * sum_roi;
    const double sq1 = sqrt((double)zn1);
    const double sq2 = sqrt((double)zn2);
    dev_result_array_bright[id] = (double)ch/ (sq1 * sq2) - (double)diff_roi_temp / ((double)sum_roi_temp);
}  // END void match_temp
#endif // END ifdef COMBINED

__global__ void max_pixel1(const float * dev_result_array_bright, Pix * max_K1)
{
    __shared__ int id, first_level_x_id, first_level_y_id, x, k, i, max_pix_x, max_pix_y;
    __shared__ float bright, max_pix_bright;
    id = blockDim.x * blockIdx.x + threadIdx.x;
    first_level_y_id = const_first_level_y[id];
    first_level_x_id = const_first_level_x[id];

    k = first_level_y_id * RESULT_WIDTH + first_level_x_id;
    bright = dev_result_array_bright[k];

    max_pix_x = first_level_x_id;
    max_pix_y = first_level_y_id;
    max_pix_bright = bright;

    for(i = 1; i < K1; ++i)
    {
        x = first_level_x_id + i;
        k = first_level_y_id * RESULT_WIDTH + x;
        bright = dev_result_array_bright[k];
        if(bright > max_pix_bright)
        {
            max_pix_x = x;
            max_pix_y = first_level_y_id;
            max_pix_bright = bright;
        } // END if(bright > max_pix.bright)
    } // END for(i = 1; i < K1; ++i)
    max_K1[id].x = max_pix_x;
    max_K1[id].y = max_pix_y;
    max_K1[id].bright = max_pix_bright;
} // END __global__ max_pixel1

__global__ void max_pixel2(const Pix * max_K1, Pix * max_K2)
{
    __shared__ int id, i, k, max_pix_x, max_pix_y;
    __shared__ float max_pix_bright;
    id = blockDim.x * blockIdx.x + threadIdx.x;
    k = id * K2;

    const Pix& max_K1_0 = max_K1[k];
    max_pix_x = max_K1_0.x;
    max_pix_y = max_K1_0.y;
    max_pix_bright = max_K1_0.bright;

    for(i = 1; i < K2; ++i)
    {
        const Pix& max_K1_i = max_K1[k + i];
        if(max_K1_i.bright > max_pix_bright)
        {
            max_pix_bright = max_K1_i.bright;
            max_pix_x = max_K1_i.x;
            max_pix_y = max_K1_i.y;
        } // END if(max_K1_i.bright > max_pix_bright)
    } // END for(i = 1; i < K2; ++i)
    max_K2[id].x = max_pix_x;
    max_K2[id].y = max_pix_y;
    max_K2[id].bright = max_pix_bright;
} // END max_pixel2

void tmml::work_cuda(const Mat& img_work, const Mat& img_temp, Pix& max_pix)
{
    img_work_gpu.upload(img_work);
    int id = 0;
    for(int y = 0; y < TEMPLATE_HEIGHT; y++)
    {
        for(int x = 0; x < TEMPLATE_WIDTH; x++)
        {
            img_temp_arr[id] = img_temp.at<unsigned char>(y, x);
            id++;
        } // END for(int x = 0; x < TEMPLATE_WIDTH; x++)
    } // END for(int y = 0; y < TEMPLATE_HEIGHT; y++)

    hipMemcpyToSymbol(HIP_SYMBOL(const_img_temp_array), img_temp_arr, sizeof(unsigned char) * TEMPLATE_AREA);


    match_temp<<<blocks_match_temp, threads_match_temp>>>(img_work_gpu, dev_result_array_bright);

// =========================================================
    max_pixel1<<<blocks_maxloc, threads_maxloc>>>(dev_result_array_bright, dev_max_K1);
    max_pixel2<<<blocks_maxloc, threads_maxloc>>>(dev_max_K1, dev_max_K2);
    hipMemcpy(max_K2, dev_max_K2, sizeof(Pix) * N2, hipMemcpyDeviceToHost);

    max_pix = max_pix0;
    for(int i = 0; i < N2; ++i)
    {
        Pix& pix_i = max_K2[i];
        if(pix_i.bright > max_pix.bright){max_pix = pix_i;}
    } // END for(int i = 0; i < N3; ++i)

    hipMemcpy(result_array_bright, dev_result_array_bright, sizeof(float) * RESULT_AREA, hipMemcpyDeviceToHost);

} // END work_cuda
