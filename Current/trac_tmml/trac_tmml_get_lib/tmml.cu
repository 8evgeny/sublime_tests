#include "hip/hip_runtime.h"
#include "tmml.hpp"

using namespace std;
using namespace cv;

__constant__ unsigned char const_img_temp_array[TEMPLATE_AREA];
__constant__ unsigned char const_first_level_x[N1];
__constant__ unsigned char const_first_level_y[N1];

void tmml::cuda_Malloc()
{
    hipMalloc((void**)&dev_result_array_x, sizeof(unsigned char) * RESULT_AREA);
    hipMalloc((void**)&dev_result_array_y, sizeof(unsigned char) * RESULT_AREA);
    hipMalloc((void**)&dev_result_array_bright, sizeof(float) * RESULT_AREA);
    Mat img_work(Size(WORK_WIDTH, WORK_HEIGHT), CV_8UC1, Scalar(0));
    img_work_gpu.upload(img_work);
    Mat img_temp(Size(TEMPLATE_WIDTH, TEMPLATE_HEIGHT), CV_8UC1, Scalar(0));
    img_temp_gpu.upload(img_temp);
    hipMalloc((void**)&dev_max_K1, sizeof(Pix) * N1);
    hipMalloc((void**)&dev_max_K2, sizeof(Pix) * N2);
} // -- END cuda_Malloc()

void tmml::cuda_Free()
{
    hipFree(&img_work_gpu);
    hipFree(&img_temp_gpu);
    hipFree(dev_result_array_x);
    hipFree(dev_result_array_y);
    hipFree(dev_result_array_bright);
    hipFree(dev_max_K1);
    hipFree(dev_max_K2);
    hipFree(const_img_temp_array);
    hipFree(const_first_level_x);
    hipFree(const_first_level_y);
} // -- END hipFree()

void tmml::fill_result_array()
{
    int id = 0;
    for(int y = 0; y < RESULT_HEIGHT; ++y)
    {
        for(int x = 0; x < RESULT_WIDTH; ++x)
        {
            result_array_x[id] = (unsigned char)x;
            result_array_y[id] = (unsigned char)y;
            id++;
        } // END for(int x = 0; x < RESULT_WIDTH; ++x)
    } // END for(int y = 0; y < RESULT_HEIGHT; ++y)
    hipMemcpy(dev_result_array_x, result_array_x, sizeof(unsigned char) * RESULT_AREA, hipMemcpyHostToDevice);
    hipMemcpy(dev_result_array_y, result_array_y, sizeof(unsigned char) * RESULT_AREA, hipMemcpyHostToDevice);
} // END fill_result_array

void tmml::fill_template_array()
{
    int id = 0;
    for(int y = 0; y < TEMPLATE_HEIGHT; ++y)
    {
        for(int x = 0; x < TEMPLATE_WIDTH; ++x)
        {
            template_array_x[id] = (unsigned char)x;
            template_array_y[id] = (unsigned char)y;
            id++;
        }  // END for(int x = 0; x < TEMPLATE_WIDTH; ++x)
    }  // END for(int y = 0; y < TEMPLATE_HEIGHT; ++y)
    hipMemcpy(dev_template_array_x, template_array_x, sizeof(unsigned char) * TEMPLATE_AREA, hipMemcpyHostToDevice);
    hipMemcpy(dev_template_array_y, template_array_y, sizeof(unsigned char) * TEMPLATE_AREA, hipMemcpyHostToDevice);
} // END fill_template_array

void tmml::fill1level()
{       
    int id = 0;
    for(int y = 0; y < RESULT_HEIGHT; ++y)
    {
        for(int x = 0; x < RESULT_WIDTH; x += K1)
        {
            first_level_x[id] = (unsigned char)x;
            first_level_y[id] = (unsigned char)y;
            id++;
        } // END for(int x = 0; x < RESULT_WIDTH; x += K1)
    } // END for(int x = 0; x < RESULT_WIDTH; x += K1)
    hipMemcpyToSymbol(HIP_SYMBOL(const_first_level_x), first_level_x, sizeof(unsigned char) * N1);
    hipMemcpyToSymbol(HIP_SYMBOL(const_first_level_y), first_level_y, sizeof(unsigned char) * N1);
} // END fill1level

#ifdef SQDIFF_NORMED
__global__ void match_temp(const cuda::PtrStepSz<unsigned char> img_work_gpu, float * dev_result_array_bright)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int sum_roi_temp = 0;
    int diff_roi_temp = 0;
    int y = id / RESULT_WIDTH;
    int x = id % RESULT_WIDTH;
    for(int tmp_y = 0; tmp_y < TEMPLATE_HEIGHT; ++tmp_y)
    {
        for(int tmp_x = 0; tmp_x < TEMPLATE_WIDTH; ++tmp_x)
        {
            int i = tmp_y * TEMPLATE_WIDTH + tmp_x;
            unsigned char temp = const_img_temp_array[i];
            unsigned char roi = img_work_gpu(tmp_y + y, tmp_x + x);
            diff_roi_temp += abs(roi - temp);
            sum_roi_temp += (roi + temp);
        } // for(int tmp_x = 0; tmp_x < TEMPLATE_WIDTH; ++tmp_x)
    } // for(int tmp_y = 0; tmp_y < TEMPLATE_HEIGHT; ++tmp_y)
    dev_result_array_bright[id] = 1.f - (float)diff_roi_temp / ((float)sum_roi_temp);
}  // END void match_temp
#endif // END #ifdef SQDIFF_NORMED

#ifdef CCOEFF_NORMED
__global__ void match_temp(const cuda::PtrStepSz<unsigned char> img_work_gpu, float * dev_result_array_bright)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int sum_roi_temp = 0;
    int sum_temp_temp = 0;
    int sum_roi_roi = 0;
    int sum_roi = 0;
    int sum_temp = 0;
    int y = id / RESULT_WIDTH;
    int x = id % RESULT_WIDTH;
    for(int tmp_y = 0; tmp_y < TEMPLATE_HEIGHT; ++tmp_y)
    {
        for(int tmp_x = 0; tmp_x < TEMPLATE_WIDTH; ++tmp_x)
        {
            int i = tmp_y * TEMPLATE_WIDTH + tmp_x;
            unsigned char temp = const_img_temp_array[i];
            unsigned char roi = img_work_gpu(tmp_y + y, tmp_x + x);
            sum_roi_temp += roi * temp;
            sum_temp_temp += temp * temp;
            sum_roi_roi += roi * roi;
            sum_roi += roi;
            sum_temp += temp;
        } // for(int tmp_x = 0; tmp_x < TEMPLATE_WIDTH; ++tmp_x)
    } // for(int tmp_y = 0; tmp_y < TEMPLATE_HEIGHT; ++tmp_y)
    const float sum_roi_temp1 = TEMPLATE_AREA_1 * sum_roi_temp;
    const float sum_roi1 = TEMPLATE_AREA_1 * sum_roi;
    const float sum_temp1 = TEMPLATE_AREA_1 * sum_temp;
    const float sum_roi_roi1 = TEMPLATE_AREA_1 * sum_roi_roi;
    const float sum_temp_temp1 = TEMPLATE_AREA_1 * sum_temp_temp;
    const float ch  = sum_roi_temp1 - sum_roi1 * sum_temp1;
    const float zn1 = sum_temp_temp1 - sum_temp1 * sum_temp1;
    const float zn2 = sum_roi_roi1 - sum_roi1 * sum_roi1;
    dev_result_array_bright[id] = ch / sqrt(zn1 * zn2);
}  // END void match_temp
#endif // END #ifdef CCOEFF_NORMED

#ifdef COMBINED
__global__ void match_temp(const cuda::PtrStepSz<unsigned char> img_work_gpu, float * dev_result_array_bright)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int sum_roi_temp = 0;
    int sum_temp_temp = 0;
    int sum_roi_roi = 0;
    int sum_roi = 0;
    int sum_temp = 0;
    int sum_roi_temp_2 = 0;
    int diff_roi_temp = 0;
    int y = id / RESULT_WIDTH;
    int x = id % RESULT_WIDTH;
    for(int tmp_y = 0; tmp_y < TEMPLATE_HEIGHT; ++tmp_y)
    {
        for(int tmp_x = 0; tmp_x < TEMPLATE_WIDTH; ++tmp_x)
        {
            int i = tmp_y * TEMPLATE_WIDTH + tmp_x;
            unsigned char temp = const_img_temp_array[i];
            unsigned char roi = img_work_gpu(tmp_y + y, tmp_x + x);
            sum_roi_temp += roi * temp;
            sum_temp_temp += temp * temp;
            sum_roi_roi += roi * roi;
            sum_roi += roi;
            sum_temp += temp;
            diff_roi_temp += abs(roi - temp);
            sum_roi_temp_2 += (roi + temp);
        } // for(int tmp_x = 0; tmp_x < TEMPLATE_WIDTH; ++tmp_x)
    } // for(int tmp_y = 0; tmp_y < TEMPLATE_HEIGHT; ++tmp_y)
    const float sum_roi_temp1 = TEMPLATE_AREA_1 * sum_roi_temp;
    const float sum_roi1 = TEMPLATE_AREA_1 * sum_roi;
    const float sum_temp1 = TEMPLATE_AREA_1 * sum_temp;
    const float sum_roi_roi1 = TEMPLATE_AREA_1 * sum_roi_roi;
    const float sum_temp_temp1 = TEMPLATE_AREA_1 * sum_temp_temp;
    const float ch  = sum_roi_temp1 - sum_roi1 * sum_temp1;
    const float zn1 = sum_temp_temp1 - sum_temp1 * sum_temp1;
    const float zn2 = sum_roi_roi1 - sum_roi1 * sum_roi1;
    dev_result_array_bright[id] = ch / sqrt(zn1 * zn2) - (float)diff_roi_temp / sum_roi_temp_2;
}  // END void match_temp
#endif // END ifdef COMBINED

__global__ void max_pixel1(const float * dev_result_array_bright, Pix * max_K1)
{
    __shared__ int id, first_level_x_id, first_level_y_id, x, k, i, max_pix_x, max_pix_y;
    __shared__ float bright, max_pix_bright;
    id = blockDim.x * blockIdx.x + threadIdx.x;
    first_level_y_id = const_first_level_y[id];
    first_level_x_id = const_first_level_x[id];

    k = first_level_y_id * RESULT_WIDTH + first_level_x_id;
    bright = dev_result_array_bright[k];

    max_pix_x = first_level_x_id;
    max_pix_y = first_level_y_id;
    max_pix_bright = bright;

    for(i = 1; i < K1; ++i)
    {
        x = first_level_x_id + i;
        k = first_level_y_id * RESULT_WIDTH + x;
        bright = dev_result_array_bright[k];
        if(bright > max_pix_bright)
        {
            max_pix_x = x;
            max_pix_y = first_level_y_id;
            max_pix_bright = bright;
        } // END if(bright > max_pix.bright)
    } // END for(i = 1; i < K1; ++i)
    max_K1[id].x = max_pix_x;
    max_K1[id].y = max_pix_y;
    max_K1[id].bright = max_pix_bright;
} // END __global__ max_pixel1

__global__ void max_pixel2(const Pix * max_K1, Pix * max_K2)
{
    __shared__ int id, i, k, max_pix_x, max_pix_y;
    __shared__ float max_pix_bright;
    id = blockDim.x * blockIdx.x + threadIdx.x;
    k = id * K2;

    const Pix& max_K1_0 = max_K1[k];
    max_pix_x = max_K1_0.x;
    max_pix_y = max_K1_0.y;
    max_pix_bright = max_K1_0.bright;

    for(i = 1; i < K2; ++i)
    {
        const Pix& max_K1_i = max_K1[k + i];
        if(max_K1_i.bright > max_pix_bright)
        {
            max_pix_bright = max_K1_i.bright;
            max_pix_x = max_K1_i.x;
            max_pix_y = max_K1_i.y;
        } // END if(max_K1_i.bright > max_pix_bright)
    } // END for(i = 1; i < K2; ++i)
    max_K2[id].x = max_pix_x;
    max_K2[id].y = max_pix_y;
    max_K2[id].bright = max_pix_bright;
} // END max_pixel2

void tmml::work_tmml(const Mat& img_work, const Mat& img_temp, Pix& max_pix)
{
    img_work_gpu.upload(img_work);
    int id = 0;
    for(int y = 0; y < TEMPLATE_HEIGHT; y++)
    {
        for(int x = 0; x < TEMPLATE_WIDTH; x++)
        {
            img_temp_arr[id] = img_temp.at<unsigned char>(y, x);
            id++;
        } // END for(int x = 0; x < TEMPLATE_WIDTH; x++)
    } // END for(int y = 0; y < TEMPLATE_HEIGHT; y++)

    hipMemcpyToSymbol(HIP_SYMBOL(const_img_temp_array), img_temp_arr, sizeof(unsigned char) * TEMPLATE_AREA);


    match_temp<<<blocks_match_temp, threads_match_temp>>>(img_work_gpu, dev_result_array_bright);

// =========================================================
    max_pixel1<<<blocks_maxloc, threads_maxloc>>>(dev_result_array_bright, dev_max_K1);
    max_pixel2<<<blocks_maxloc, threads_maxloc>>>(dev_max_K1, dev_max_K2);
    hipMemcpy(max_K2, dev_max_K2, sizeof(Pix) * N2, hipMemcpyDeviceToHost);

    max_pix = max_pix0;
    for(int i = 0; i < N2; ++i)
    {
        Pix& pix_i = max_K2[i];
        if(pix_i.bright > max_pix.bright){max_pix = pix_i;}
    } // END for(int i = 0; i < N3; ++i)

#ifdef find_diff_result
    hipMemcpy(result_array_bright, dev_result_array_bright, sizeof(float) * RESULT_AREA, hipMemcpyDeviceToHost);
#endif // END #ifdef find_diff_result
} // END work_tmml
